#include <zisa/memory/copy_bytes.hpp>

namespace zisa {

#if ZISA_HAS_CUDA == 1
void copy_bytes(void * dst, const device_type &dst_loc, void * src, const device_type &src_loc, std::size_t n_bytes) {
  if(src_loc == device_type::cpu && dst_loc == device_type::cpu) {
    hipMemcpy(dst, src, n_bytes, hipMemcpyHostToHost);
  }
  else if(src_loc == device_type::cpu && dst_loc == device_type::cuda) {
    hipMemcpy(dst, src, n_bytes, hipMemcpyHostToDevice);
  }
  else if(src_loc == device_type::cuda && dst_loc == device_type::cpu) {
    hipMemcpy(dst, src, n_bytes, hipMemcpyDeviceToHost);
  }
  else if(src_loc == device_type::cuda && dst_loc == device_type::cuda) {
    hipMemcpy(dst, src, n_bytes, hipMemcpyDeviceToDevice);
  }
  else {
    LOG_ERR("Unknown combination of `device_type`.");
  }
}
#endif

}
