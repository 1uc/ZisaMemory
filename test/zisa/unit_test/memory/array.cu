#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (c) 2021 ETH Zurich, Luc Grosheintz-Laval

#include <zisa/memory/array.hpp>
#include <zisa/memory/array_view.hpp>
#include <zisa/memory/column_major.hpp>
#include <zisa/testing/testing_framework.hpp>

using namespace zisa;

__global__ void call_kernel(array_view<double, 2> y, array<double, 2> x) {
  if (threadIdx.x == 0) {
    x(0, 0) = 42.0;
    y(0, 0) = x(0, 0);
  }
}

template <int NDIMS>
static bool f(const array_const_view<double, NDIMS> &) {
  return true;
}

TEST_CASE("array/array_view; API", "[cuda][array]") {
  auto a = array<double, 2>({3, 2}, device_type::cuda);
  auto b = array<double, 2>({3, 2}, device_type::cuda);

  auto b_view = array_view<double, 2>(b);
  call_kernel<<<1, 1>>>(b_view, a);
  call_kernel<<<1, 1>>>(b.view(), a);

  // f(b);  // template deduction fails.
  f(b.const_view());
}
